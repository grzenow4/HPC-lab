#include "hip/hip_runtime.h"
#include "./common/helpers.h"

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x + blockDim.x;
    if (tid < N) {
        int tid1 = (tid + 1) % 256;
        int tid2 = (tid + 2) % 256;
        float aSum = (a[tid] + a[tid1] + a[tid2]) / 3.0f;
        float bSum = (b[tid] + b[tid1] + b[tid2]) / 3.0f;
        c[tid] = (aSum + bSum) / 2;
    }
}

int main(void) {
    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    int *host_a, *host_b, *host_c;
    int *dev_a1, *dev_b1, *dev_c1;
    int *dev_a2, *dev_b2, *dev_c2;

    HANDLE_ERROR(hipMalloc((void**)&dev_a1, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b1, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c1, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_a2, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b2, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c2, N * sizeof(int)));

    HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    for (int i = 0; i < FULL_DATA_SIZE / 2; i += N) {
        HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a + 2 * i, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        HANDLE_ERROR(hipMemcpyAsync(dev_b1, host_b + 2 * i, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        HANDLE_ERROR(hipMemcpyAsync(dev_a2, host_a + 2 * i + N, N * sizeof(int), hipMemcpyHostToDevice, stream2));
        HANDLE_ERROR(hipMemcpyAsync(dev_b2, host_b + 2 * i + N, N * sizeof(int), hipMemcpyHostToDevice, stream2));

        kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
        kernel<<<N / 256, 256, 0, stream2>>>(dev_a2, dev_b2, dev_c2);

        HANDLE_ERROR(hipMemcpyAsync(host_c + 2 * i, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
        HANDLE_ERROR(hipMemcpyAsync(host_c + 2 * i + N, dev_c2, N * sizeof(int), hipMemcpyDeviceToHost, stream2));
    }

    HANDLE_ERROR(hipStreamSynchronize(stream1));
    HANDLE_ERROR(hipStreamSynchronize(stream2));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a1));
    HANDLE_ERROR(hipFree(dev_b1));
    HANDLE_ERROR(hipFree(dev_c1));
    HANDLE_ERROR(hipFree(dev_a2));
    HANDLE_ERROR(hipFree(dev_b2));
    HANDLE_ERROR(hipFree(dev_c2));

    HANDLE_ERROR(hipStreamDestroy(stream1));
    HANDLE_ERROR(hipStreamDestroy(stream2));

    return 0;
}
